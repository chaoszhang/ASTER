#include "hip/hip_runtime.h"
#include "biallelic-cuda.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__constant__ short TAPE[MAX_TAPE_SIZE * 4];

__global__ void updateBatch(GroupIBatch *GI, SiteBatch *S, HotBatch *H, int n, int x, int y, int i){
	int tx = threadIdx.x;
    int bx = blockIdx.x;

	int len = GI[bx][1] - GI[bx][0];
	int start = n * GI[bx][0] + i * len;
	
	for (int j = 0; j < len; j++){
		for (int p = 0; p < 4; p++){
			if (y != -1) S[GI[bx][0] + j][p][y][tx] -= ((H[start + j][p] >> tx) & 1);
			if (x != -1) S[GI[bx][0] + j][p][x][tx] += ((H[start + j][p] >> tx) & 1);
		}
	}
}

void cudaUpdate(GroupIBatch *GI, SiteBatch *S, HotBatch *H, int n, int x, int y, int i, int kc, int b){
	updateBatch<<<kc, b>>>(GI, S, H, n, x, y, i);
}

__global__ void scoreBatch(GroupIBatch *GI, GroupFBatch *GF, SiteBatch *S, double *BS){
	int tx = threadIdx.x;
    int bx = blockIdx.x;

	__shared__ int sXX[7][BATCH], sYZ[7][BATCH], sXpXq[7][BATCH], sYpZq[7][BATCH];
	__shared__ int sYY[7][BATCH], sXZ[7][BATCH], sYpYq[7][BATCH], sXpZq[7][BATCH];
	__shared__ int sZZ[7][BATCH], sXY[7][BATCH], sZpZq[7][BATCH], sXpYq[7][BATCH];
	__shared__ long long m[7][BATCH], mpq[7][BATCH], mp2q2[7][BATCH];
	
	for (int p = 0; p < 7; p++){
		sXX[p][tx] = 0; sYZ[p][tx] = 0; sXpXq[p][tx] = 0; sYpZq[p][tx] = 0;
		sYY[p][tx] = 0; sXZ[p][tx] = 0; sYpYq[p][tx] = 0; sXpZq[p][tx] = 0;
		sZZ[p][tx] = 0; sXY[p][tx] = 0; sZpZq[p][tx] = 0; sXpYq[p][tx] = 0;
		m[p][tx] = 0; mpq[p][tx] = 0; mp2q2[p][tx] = 0;
	}
	for (int j = GI[bx][0], jMax = GI[bx][1]; j < jMax; j++){
		int x = 0, y = 0, z = 0;
		short (&Xh)[4][3][BATCH] = S[j];
		for (int p = 0; p < 4; p++){
			x += Xh[p][0][tx];
			y += Xh[p][1][tx];
			z += Xh[p][2][tx];
		}
		for (int p = 0; p < 7; p++){
			int xp = (p < 4) ? Xh[p][0][tx] : Xh[0][0][tx] + Xh[p-3][0][tx];
			int yp = (p < 4) ? Xh[p][1][tx] : Xh[0][1][tx] + Xh[p-3][1][tx];
			int zp = (p < 4) ? Xh[p][2][tx] : Xh[0][2][tx] + Xh[p-3][2][tx];
			const int xq = x - xp, yq = y - yp, zq = z - zp;
			
			const int xx = x * (x - 1), xpxq = xp * xq, yz = y * z, ypzq = yp * zq + zp * yq;
			const int yy = y * (y - 1), ypyq = yp * yq, xz = x * z, xpzq = xp * zq + zp * xq;
			const int zz = z * (z - 1), zpzq = zp * zq, xy = x * y, xpyq = xp * yq + yp * xq;
			
			sXX[p][tx] += xx; sXpXq[p][tx] += xpxq; sYZ[p][tx] += yz; sYpZq[p][tx] += ypzq;
			sYY[p][tx] += yy; sYpYq[p][tx] += ypyq; sXZ[p][tx] += xz; sXpZq[p][tx] += xpzq;
			sZZ[p][tx] += zz; sZpZq[p][tx] += zpzq; sXY[p][tx] += xy; sXpYq[p][tx] += xpyq;
			
			const long long _xx = xx, _xpxq = xpxq, _yz = yz, _ypzq = ypzq;
			const long long _yy = yy, _ypyq = ypyq, _xz = xz, _xpzq = xpzq;
			const long long _zz = zz, _zpzq = zpzq, _xy = xy, _xpyq = xpyq;

			m[p][tx] += _xpxq * _ypzq + _ypyq * _xpzq + _zpzq * _xpyq;
			mpq[p][tx] -= 2 * (_xpxq * _yz + _ypyq * _xz + _zpzq * _xy)
				 		+ _xx * _ypzq + _yy * _xpzq + _zz * _xpyq;
			mp2q2[p][tx] += 2 * (_xx * _yz + _yy * _xz + _zz * _xy);
		}
	}
	double result = 0.0;
	for (int p = 0; p < 7; p++){
		const double pv = (p < 4) ? GF[bx][p][tx] : GF[bx][0][tx] + GF[bx][p-3][tx]; 
		const double pq = pv * (1 - pv);
		result += (pq * sXX[p][tx] - sXpXq[p][tx]) * (2 * pq * sYZ[p][tx] - sYpZq[p][tx])
				+ (pq * sYY[p][tx] - sYpYq[p][tx]) * (2 * pq * sXZ[p][tx] - sXpZq[p][tx])
				+ (pq * sZZ[p][tx] - sZpZq[p][tx]) * (2 * pq * sXY[p][tx] - sXpYq[p][tx])
				- ((mp2q2[p][tx] * pq + mpq[p][tx]) * pq + m[p][tx]);
	}
	__shared__ double results[BATCH];
	results[tx] = result * GF[bx][4][tx];
	for (int b = 2; b <= BATCH && (tx&(b-1)) == 0; b *= 2){
		results[tx] += results[tx + b/2];
	}
	if (tx == 0) BS[bx] = results[tx];
}

void cudaScore(GroupIBatch *GI, GroupFBatch *GF, SiteBatch* S, double* BS, int kc, int b){
	scoreBatch<<<kc, b>>>(GI, GF, S, BS);
}

void myCudaMalloc(void **p, int size){
	hipMalloc(p, size);
}

void myCudaMemcpyH2D(void *T, void *S, int size){
	hipMemcpy(T, S, size, hipMemcpyHostToDevice);
}

void myCudaMemcpyD2H(void *T, void *S, int size){
	hipMemcpy(T, S, size, hipMemcpyDeviceToHost);
}

void myCudaFree(void *p){
	hipFree(p);
}

void myCudaMemcpyH2C(void *S, int size){
	hipMemcpyToSymbol(HIP_SYMBOL(TAPE), S, size);
}

__global__ void cudaWorkBatch(GroupIBatch *GI, GroupFBatch *GF, SiteBatch* S, double* BS, HotBatch *H, int n, int tapeSize){
	int tx = threadIdx.x;
    int bx = blockIdx.x;
	int gs = gridDim.x;

	const int jStart = GI[bx][0], jEnd = GI[bx][1];
	const int len = jEnd - jStart;

	__shared__ int sXX[7][BATCH], sYZ[7][BATCH], sXpXq[7][BATCH], sYpZq[7][BATCH];
	__shared__ int sYY[7][BATCH], sXZ[7][BATCH], sYpYq[7][BATCH], sXpZq[7][BATCH];
	__shared__ int sZZ[7][BATCH], sXY[7][BATCH], sZpZq[7][BATCH], sXpYq[7][BATCH];
	__shared__ long long m[7][BATCH], mpq[7][BATCH], mp2q2[7][BATCH];

	for (int r = 0, s = bx; r < tapeSize; r+=4){
		{
			int x = TAPE[r], y = TAPE[r+1], i = TAPE[r+2];
			int start = n * jStart + i * len;
			for (int j = 0; j < len; j++){
				for (int p = 0; p < 4; p++){
					if (y != -1) S[jStart + j][p][y][tx] -= ((H[start + j][p] >> tx) & 1);
					if (x != -1) S[jStart + j][p][x][tx] += ((H[start + j][p] >> tx) & 1);
				}
			}
		}
		if (TAPE[r+3] > 0){
			for (int p = 0; p < 7; p++){
				sXX[p][tx] = 0; sYZ[p][tx] = 0; sXpXq[p][tx] = 0; sYpZq[p][tx] = 0;
				sYY[p][tx] = 0; sXZ[p][tx] = 0; sYpYq[p][tx] = 0; sXpZq[p][tx] = 0;
				sZZ[p][tx] = 0; sXY[p][tx] = 0; sZpZq[p][tx] = 0; sXpYq[p][tx] = 0;
				m[p][tx] = 0; mpq[p][tx] = 0; mp2q2[p][tx] = 0;
			}
			for (int j = jStart; j < jEnd; j++){
				int x = 0, y = 0, z = 0;
				short (&Xh)[4][3][BATCH] = S[j];
				for (int p = 0; p < 4; p++){
					x += Xh[p][0][tx];
					y += Xh[p][1][tx];
					z += Xh[p][2][tx];
				}
				for (int p = 0; p < 7; p++){
					int xp = (p < 4) ? Xh[p][0][tx] : Xh[0][0][tx] + Xh[p-3][0][tx];
					int yp = (p < 4) ? Xh[p][1][tx] : Xh[0][1][tx] + Xh[p-3][1][tx];
					int zp = (p < 4) ? Xh[p][2][tx] : Xh[0][2][tx] + Xh[p-3][2][tx];
					const int xq = x - xp, yq = y - yp, zq = z - zp;
					
					const int xx = x * (x - 1), xpxq = xp * xq, yz = y * z, ypzq = yp * zq + zp * yq;
					const int yy = y * (y - 1), ypyq = yp * yq, xz = x * z, xpzq = xp * zq + zp * xq;
					const int zz = z * (z - 1), zpzq = zp * zq, xy = x * y, xpyq = xp * yq + yp * xq;
					
					sXX[p][tx] += xx; sXpXq[p][tx] += xpxq; sYZ[p][tx] += yz; sYpZq[p][tx] += ypzq;
					sYY[p][tx] += yy; sYpYq[p][tx] += ypyq; sXZ[p][tx] += xz; sXpZq[p][tx] += xpzq;
					sZZ[p][tx] += zz; sZpZq[p][tx] += zpzq; sXY[p][tx] += xy; sXpYq[p][tx] += xpyq;
					
					const long long _xx = xx, _xpxq = xpxq, _yz = yz, _ypzq = ypzq;
					const long long _yy = yy, _ypyq = ypyq, _xz = xz, _xpzq = xpzq;
					const long long _zz = zz, _zpzq = zpzq, _xy = xy, _xpyq = xpyq;

					m[p][tx] += _xpxq * _ypzq + _ypyq * _xpzq + _zpzq * _xpyq;
					mpq[p][tx] -= 2 * (_xpxq * _yz + _ypyq * _xz + _zpzq * _xy)
								+ _xx * _ypzq + _yy * _xpzq + _zz * _xpyq;
					mp2q2[p][tx] += 2 * (_xx * _yz + _yy * _xz + _zz * _xy);
				}
			}
			double result = 0.0;
			for (int p = 0; p < 7; p++){
				const double pv = (p < 4) ? GF[bx][p][tx] : GF[bx][0][tx] + GF[bx][p-3][tx]; 
				const double pq = pv * (1 - pv);
				result += (pq * sXX[p][tx] - sXpXq[p][tx]) * (2 * pq * sYZ[p][tx] - sYpZq[p][tx])
						+ (pq * sYY[p][tx] - sYpYq[p][tx]) * (2 * pq * sXZ[p][tx] - sXpZq[p][tx])
						+ (pq * sZZ[p][tx] - sZpZq[p][tx]) * (2 * pq * sXY[p][tx] - sXpYq[p][tx])
						- ((mp2q2[p][tx] * pq + mpq[p][tx]) * pq + m[p][tx]);
			}
			__shared__ double results[BATCH];
			results[tx] = result * GF[bx][4][tx];
			for (int b = 2; b <= BATCH && (tx&(b-1)) == 0; b *= 2){
				results[tx] += results[tx + b/2];
			}
			if (tx == 0) BS[s] = results[tx];
			s += gs;
		}
	}
}

void cudaWork(GroupIBatch *GI, GroupFBatch *GF, SiteBatch* S, double* BS, HotBatch *H, int n, int kc, int b, int tapeSize){
	cudaWorkBatch<<<kc, b>>>(GI, GF, S, BS, H, n, tapeSize);
}
